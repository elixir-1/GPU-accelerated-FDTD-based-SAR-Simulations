#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "parallel_var.h"
#include <iostream>
#include <thrust/device_vector.h>

using namespace std;

// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }

#define pi 4.0*atan(1.0)

__global__ void zero_fields(double *EXS, double *EYS, double *EZS, double *HXS, double *HYS, double *HZS, int *IDONE, int *IDTWO, int *IDTHREE, int N){
  
  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  if(tid<N){
    EXS[tid]=0.0;
    EYS[tid]=0.0;
    EZS[tid]=0.0;
    HXS[tid]=0.0;
    HYS[tid]=0.0;
    HZS[tid]=0.0;
    IDONE[tid]=0;
    IDTWO[tid]=0;
    IDTHREE[tid]=0;
  }

}

__global__ void zero_xplanes(double *EYSX1, double *EYSX2, double *EZSX1, double *EZSX2, int N){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  if(tid<N){
    EYSX1[tid]=0.0;
    EYSX2[tid]=0.0;
    EZSX1[tid]=0.0;
    EZSX2[tid]=0.0;
  }

}

__global__ void zero_yplanes(double *EXSY1, double *EXSY2, double *EZSY1, double *EZSY2, int N){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  if(tid<N){
    EXSY1[tid]=0.0;
    EXSY2[tid]=0.0;
    EZSY1[tid]=0.0;
    EZSY2[tid]=0.0;
  }

}

__global__ void zero_zplanes(double *EXSZ1, double *EXSZ2, double *EYSZ1, double *EYSZ2, int N){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  if(tid<N){
    EYSZ1[tid]=0.0;
    EYSZ2[tid]=0.0;
    EXSZ1[tid]=0.0;
    EXSZ2[tid]=0.0;
  }
  return;
}

__global__ void zeromdim(void){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  if(tid<mdim1){
    ESCTC[tid]=0.0;
    EINCC[tid]=0.0;
    EDEVCN[tid]=0.0;
    ECRLX[tid]=0.0;
    ECRLY[tid]=0.0;
    ECRLZ[tid]=0.0;
  }

  return;
}

__host__ __device__ long long int index(int i,int j, int k, int x, int y, int z){
  return (i*(y*z)+j*z+k);
}

__device__ void dcube(int **IDONE, int **IDTWO, int **IDTHREE, int istart, int jstart, int kstart, int nxwide, int nywide, int nzwide, int mtype){

  int imax, jmax, kmax;
  int i,j,k;
  
  imax = istart+nxwide-1;
  jmax = jstart+nywide-1;
  kmax = kstart+nzwide-1;

  if(nxwide==0){
    for(k=kstart;k<=kmax;k++){
      for(j=jstart;j<=jmax;j++){
	(*IDTWO)[index(istart,j,k,NX,NY,NZ)] = mtype;
	(*IDTWO)[index(istart,j,k+1,NX,NY,NZ)] = mtype;
	(*IDTHREE)[index(istart,j,k,NX,NY,NZ)] = mtype;
	(*IDTHREE)[index(istart,j,k+1,NX,NY,NZ)] = mtype;
      }
    }
  }

  else if(nywide==0){
    for(k=kstart;k<=kmax;k++){
      for(i=istart;i<=imax;i++){
	(*IDONE)[jstart+i*NY+k*(NX*NY)] = mtype;
	(*IDONE)[jstart+i*NY+(k+1)*(NX*NY)] = mtype;
	(*IDTHREE)[jstart+i*NY+k*(NX*NY)] = mtype;
	(*IDTHREE)[jstart+i*NY+(k+1)*(NX*NY)] = mtype;
      }
    }
  }

  else if(nzwide==0){
    for(j=jstart;j<=jmax;j++){
      for(i=istart;i<=imax;i++){
	(*IDONE)[index(i,j,kstart,NX,NY,NZ)] = mtype;
	(*IDONE)[index(i,j,kstart+1,NX,NY,NZ)] = mtype;
	(*IDTWO)[index(i,j,kstart,NX,NY,NZ)] = mtype;
	(*IDTWO)[index(i,j,kstart+1,NX,NY,NZ)] = mtype;
      }
    }
  }

  else{
    for(k=kstart;k<=kmax;k++){
      for(j=jstart;j<=jmax;j++){
	for(i=istart;i<=imax;i++){
	  (*IDONE)[index(i,j,k,NX,NY,NZ)] = mtype;
	  (*IDONE)[index(i,j,k+1,NX,NY,NZ)] = mtype;
	  (*IDONE)[index(i,j+1,k+1,NX,NY,NZ)] = mtype;
	  (*IDONE)[index(i,j+1,k,NX,NY,NZ)] = mtype;
	  (*IDTWO)[index(i,j,k,NX,NY,NZ)] = mtype;
	  (*IDTWO)[index(i+1,j,k,NX,NY,NZ)] = mtype;
	  (*IDTWO)[index(i+1,j,k+1,NX,NY,NZ)] = mtype;
	  (*IDTWO)[index(i,j,k+1,NX,NY,NZ)] = mtype;
	  (*IDTHREE)[index(i,j,k,NX,NY,NZ)] = mtype;
	  (*IDTHREE)[index(i+1,j,k,NX,NY,NZ)] = mtype;
	  (*IDTHREE)[index(i+1,j+1,k,NX,NY,NZ)] = mtype;
	  (*IDTHREE)[index(i,j+1,k,NX,NY,NZ)] = mtype;
	}
      }
    }
  }
  return;
  
}

__global__ void build(int NX, int NY, int NZ, int *IDONE, int *IDTWO, int *IDTHREE, long N){
  
  long tid=blockIdx.x*blockDim.x+threadIdx.x;
  
  if(tid<N){
    int mtype=2;
    double temp;
    double r2=0.5, r3=0.1;

    int i,j,k;

    k=(tid%(NY*NZ))%NZ;
    j=(tid%(NY*NZ))/NZ;
    i=tid/(NY*NZ);
    
    nxc=((NX+1)/2)-1;
    nyc=((NY+1)/2)-1;
    nzc=((NZ+1)/2)-1;

    temp=((pow((i-nxc)*delx,2)/pow(r3,2)) + (pow((j-nyc)*dely,2)/pow(r3,2)) + (pow((k-nzc)*delz,2)/pow(r2,2)));
    
    //temp=(pow((i-nxc)*delx,2) + pow((j-nyc)*dely,2) + pow((k-nzc)*delz,2));
    // r1=sqrt(temp);
    
    // if(radius2<delx || radius2<dely || radius2<delz)
    //   exit(-1);

    // if(r1<=radius2){
    //   dcube(&IDONE,&IDTWO,&IDTHREE,i,j,k,1,1,1,mtype);
    // }

    if(temp>0.0 && temp<=1.0){
      dcube(&IDONE,&IDTWO,&IDTHREE,i,j,k,1,1,1,mtype);
    }
  }
  return;
}

__global__ void setup(double *dt)
{

  // THIS SUBROUTINE INITIALIZES THE COMPUTATIONS
  double dtxi, dtyi, dtzi;
  int i;

  dtxi = c/delx;
  dtyi = c/dely;
  dtzi = c/delz;

  // CALCULATE DT--THE MAXIMUM TIME STEP ALLOWED BY THE COURANT STABILITY CONDITION
  *dt = (1.0/sqrt(dtxi*dtxi+dtyi*dtyi+dtzi*dtzi));
	
  // 	PARAMETER ALPHA IS THE DECAY RATE DETERMINED BY BETA.
  // TO CHANGE THE GAUSSIAN PULSE BY SINE WAVE WE HAVE TO MODIFY THE
  // CODE .WHERE EVER WE ARE USING THE ALPHA AND BETA WE HAVE
  // TO REMOVE IT .AND ALSO WE HAVE TO CHANGE THE TIME DURATION
  // ALSO TO EXIT THE SINE WAVE FOR THAT INTERVAL OF TIME.
	

  // in 3d fortran parameters.h
  period = 1e-6;
  
  // SET OFFSET FOR COMPUTING INCIDENT FIELDS

  off = 10;

  // THE FOLLOWING LINES ARE FOR SMOOTH COSINE INCIDENT FUNCTION
  // FIND DIRECTION COSINES FOR INCIDENT FIELD

  double costh = cos(pi*thinc/180.0);
  double sinth = sin(pi*thinc/180.0);
  double cosph = cos(pi*phinc/180.0);
  double sinph = sin(pi*phinc/180.0);
	
  // FIND AMPLITUDE OF INCIDENT FIELD COMPONENTS
	
  ampx = amp*(ethinc*cosph*costh - ephinc*sinph);
  ampy = amp*(ethinc*sinph*costh + ephinc*cosph);
  ampz = amp*(-ethinc*sinth);

  //printf("%lf %lf %lf\n",ampx,ampy,ampz);
  // FIND RELATIVE SPATIAL DELAY FOR X, Y, Z CELL DISPLACEMENT

  xdisp = -cosph*sinth;
  ydisp = -sinth*sinph;
  zdisp = -costh;
  //printf("%.15lf %.15lf %.15lf\n",xdisp,ydisp,zdisp);
  // for(i=1;i<mdim1;i++){
  //   EPS[i] = 0.0;
  //   SIGMA[i]=0.0;
  // }

  for(i=1;i<mdim1;i++){
    Ep[i]=45.5;
  }
  
  for(i = 1; i < mdim1; i++){
    EPS[i] = Ep[i]*eps0;
    SIGMA[i] = 1.75; 
    //printf("%.15lf %.15lf\n",EPS[i],SIGMA[i]);
  }
  
  // FREE SPACE:
  
  dtedx = (*dt)/(eps0*delx);
  dtedy = (*dt)/(eps0*dely);
  dtedz = (*dt)/(eps0*delz);
  dtmdx = (*dt)/(xmu0*delx);
  dtmdy = (*dt)/(xmu0*dely);
  dtmdz = (*dt)/(xmu0*delz);
  // lossy dielectrics
  
  for(i = 1; i < mdim1; i++)
    {
      ESCTC[i] = EPS[i]/(EPS[i]+SIGMA[i]*(*dt));
      EINCC[i] = SIGMA[i] *(*dt)/(EPS[i]+SIGMA[i]*(*dt));
      EDEVCN[i] = (*dt)*(EPS[i]-eps0)/(EPS[i]+SIGMA[i]*(*dt));
      ECRLX[i] = (*dt)/((EPS[i]+SIGMA[i]*(*dt))*delx);
      ECRLY[i] = (*dt)/((EPS[i]+SIGMA[i]*(*dt))*dely);
      ECRLZ[i] = (*dt)/((EPS[i]+SIGMA[i]*(*dt))*delz);
      //printf("%.15lf %.15lf %.15lf %.15lf %.15lf %.15lf\n",ESCTC[i],EINCC[i],EDEVCN[i],ECRLX[i],ECRLY[i],ECRLZ[i]);
    }

  // FIND MAXIMUM SPATIAL DELAY TO MAKE SURE PULSE PROPAGATES INTO SPACE PROPERLY.
  
  delay = 0.0;
  
  if(xdisp < 0.0)
    delay = delay-(xdisp*NX1*delx);
  if(ydisp < 0.0)
    delay = delay-(ydisp*NY1*dely);
  if(zdisp < 0.0)
    delay = delay-(zdisp*NZ1*delz);

  //printf("delay: %.25lf\n",delay);

  // COMPUTE OUTER RADIATION BOUNDARY CONDITION (ORBC) CONSTANTS:
  cxd =(c*(*dt)-delx)/(c*(*dt)+delx);
  cyd =(c*(*dt)-dely)/(c*(*dt)+dely);
  czd =(c*(*dt)-delz)/(c*(*dt)+delz);

  cxu = cxd;
  cyu = cyd;
  czu = czd;

  // COMPUTE 2ND ORDER ORBC CONSTANTS
  cxx = 2.0*delx/(c*(*dt)+delx);
  cyy = 2.0*dely/(c*(*dt)+dely);
  czz = 2.0*delz/(c*(*dt)+delz);

  cxfyd = delx*c*(*dt)*c*(*dt)/(2.0*dely*dely*(c*(*dt)+delx));
  cxfzd = delx*c*(*dt)*c*(*dt)/(2.0*delz*delz*(c*(*dt)+delx));
  cyfzd = dely*c*(*dt)*c*(*dt)/(2.0*delz*delz*(c*(*dt)+dely));
  cyfxd = dely*c*(*dt)*c*(*dt)/(2.0*delx*delx*(c*(*dt)+dely));
  czfxd = delz*c*(*dt)*c*(*dt)/(2.0*delx*delx*(c*(*dt)+delz));
  czfyd = delz*c*(*dt)*c*(*dt)/(2.0*dely*dely*(c*(*dt)+delz));	

}


__device__ double source(double dist, double t)
{
  double sourcev=0.0;
  tau=t-dist/c;
  if(tau<0.0)
    return sourcev;
  if(tau>period)
    return sourcev;
  double omega=2.0*pi*freq;
  sourcev=sin(omega*tau);
  return sourcev;
}

__device__ double dsrce(double dist, double t)
{
  double dsrcev;
  dsrcev=0.0;
  tau=t-dist/c;
  if(tau<0.0)
    return dsrcev;
  if(tau>period)
    return dsrcev;
  double omega=2.0*pi*freq;
  dsrcev=cos(omega*tau)*omega;
  return dsrcev;
}

__device__ double EXI(int i, int j, int k, double t){

  double dist;
  dist = ((i)*delx+0.5*delx*off)*xdisp+((j)*dely)*ydisp+((k)*delz)*zdisp + delay;
  double s = source(dist,t);
  return ampx*s;
}

__device__ double EYI(int i, int j, int k, double t)
{
  double dist;

  dist = ((i)*delx)*xdisp+((j)*dely+0.5*dely*off)*ydisp+((k)*delz)*zdisp + delay;

  return ampy*source(dist,t);
}

__device__ double EZI(int i, int j, int k, double t)
{
  double dist;

  dist = ((i)*delx)*xdisp+((j)*dely)*ydisp+((k)*delz+0.5*delz*off)*zdisp + delay;

  return ampz*source(dist,t);
}

__device__ double DEXI(int i, int j, int k, double t)
{
  double dist;

  dist = ((i)*delx+0.5*delx*off)*xdisp+((j)*dely)*ydisp+((k)*delz)*zdisp + delay;

  return ampx*dsrce(dist,t);
}

__device__ double DEYI(int i, int j, int k, double t)
{
  double dist;

  dist = ((i)*delx)*xdisp+((j)*dely+0.5*dely*off)*ydisp+((k)*delz)*zdisp+delay;

  return ampy*dsrce(dist,t);
}

__device__ double DEZI(int i, int j, int k, double t)
{
  double dist;

  dist = ((i)*delx)*xdisp+((j)*dely*off)*ydisp+((k)*delz+0.5*delz*off)*zdisp+delay;

  return ampz*dsrce(dist,t);
}

__global__ void eupdate(double *EXS, double *EYS, double *EZS, double *HXS, double *HYS, double *HZS, int *IDONE, int *IDTWO, int *IDTHREE, int NZ1, int NY1, int NX1, double t){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(k>0 && k<NZ1 && j>0 && j<NY1 && i<NX1){

    if(IDONE[tid]==0){
      EXS[tid]=EXS[tid]+(HZS[tid]-HZS[index(i,j-1,k,NX,NY,NZ)])*dtedy-(HYS[tid]-HYS[index(i,j,k-1,NX,NY,NZ)])*dtedz;
    }

    else if(IDONE[tid]==1){
      EXS[tid] = -EXI(i,j,k,t);
    }

    else if(IDONE[tid] == 2 || IDONE[tid] == 3 || IDONE[tid] == 4 || IDONE[tid] == 5){
      EXS[tid]=EXS[tid]*ESCTC[IDONE[tid]-1]-EINCC[IDONE[tid]-1]*EXI(i,j,k,t)-EDEVCN[IDONE[tid]-1]*DEXI(i,j,k,t)+(HZS[tid]-HZS[index(i,j-1,k,NX,NY,NZ)])*ECRLY[IDONE[tid]-1]-(HYS[tid]-HYS[index(i,j,k-1,NX,NY,NZ)])*ECRLZ[IDONE[tid]-1];
    }
    
  }

  if(k>0 && k<NZ1 && j<NY1 && i>0 && i<NX1){

    if(IDTWO[tid]==0){
      EYS[tid] = EYS[tid]+(HXS[tid]-HXS[index(i,j,k-1,NX,NY,NZ)])*dtedz-(HZS[tid]-HZS[index(i-1,j,k,NX,NY,NZ)])*dtedx;
    }

    else if(IDTWO[tid]==1){
      EYS[tid] = -EYI(i,j,k,t);
    }

    else if(IDTWO[tid]==2 || IDTWO[tid]==3 || IDTWO[tid]==4 || IDTWO[tid]==5){
      EYS[tid]=EYS[tid]*ESCTC[IDTWO[tid]-1]-EINCC[IDTWO[tid]-1]*EYI(i,j,k,t)-EDEVCN[IDTWO[tid]-1]*DEYI(i,j,k,t)+(HXS[tid]-HXS[index(i,j,k-1,NX,NY,NZ)])*ECRLZ[IDTWO[tid]-1]-(HZS[tid]-HZS[index(i-1,j,k,NX,NY,NZ)])*ECRLX[IDTWO[tid]-1];
    }
    
  }

  if(k<NZ1 && j>0 && j<NY1 && i>0 && i<NX1){

    if(IDTHREE[tid]==0){
      EZS[tid] = EZS[tid]+(HYS[tid]-HYS[index(i-1,j,k,NX,NY,NZ)])*dtedx-(HXS[tid]-HXS[index(i,j-1,k,NX,NY,NZ)])*dtedy;
    }

    else if(IDTHREE[tid]==1){
      EZS[tid] = -EZI(i,j,k,t);
    }

    else if(IDTHREE[tid]==2 || IDTHREE[tid]==3 || IDTHREE[tid]==4 || IDTHREE[tid]==5){
      EZS[tid]=EZS[tid]*ESCTC[IDTHREE[tid]-1]-EINCC[IDTHREE[tid]-1]*EZI(i,j,k,t)-EDEVCN[IDTHREE[tid]-1]*DEZI(i,j,k,t)+(HYS[tid]-HYS[index(i-1,j,k,NX,NY,NZ)])*ECRLX[IDTHREE[tid]-1]-(HXS[tid]-HXS[index(i,j-1,k,NX,NY,NZ)])*ECRLY[IDTHREE[tid]-1];
    }
    
  }
}

__global__ void radezx1(double *EZS, double *EZSX1, double *EZSX2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(i==0 && k<NZ1){
    j=1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(1,j,k,4,NY1,NZ1)]+cxd*(EZS[index(1,j,k,NX,NY,NZ)]-EZSX1[index(i,j,k,4,NY1,NZ1)]);
    j = NY1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EZS[index(1,j,k,NX,NY,NZ)] - EZSX1[index(i,j,k,4,NY1,NZ1)]);
  }

  if(i==NX-1 && k<NZ1){
    j=1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EZS[index(NX1-1,j,k,NX,NY,NZ)] - EZSX1[index(3,j,k,4,NY1,NZ1)]);
    j = NY1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EZS[index(NX1-1,j,k,NX,NY,NZ)] - EZSX1[index(3,j,k,4,NY1,NZ1)]);
  }

  if(i==0 && j>1 && j<NY1-1){
    k=0;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EZS[index(1,j,k,NX,NY,NZ)] - EZSX1[index(i,j,k,4,NY1,NZ1)]);
    k = NZ1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EZS[index(1,j,k,NX,NY,NZ)] - EZSX1[index(i,j,k,4,NY1,NZ1)]);
  }

  if(i==NX-1 && j>1 && j<NY1-1){
    k=0;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EZS[index(NX1-1,j,k,NX,NY,NZ)] - EZSX1[index(3,j,k,4,NY1,NZ1)]);
    k = NZ1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EZS[index(NX1-1,j,k,NX,NY,NZ)] - EZSX1[index(3,j,k,4,NY1,NZ1)]);
  }
}

__global__ void radezx2(double *EZS, double *EZSX1, double *EZSX2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(i==0 && k>0 && k<NZ1-1 && j>1 && j<NY1-1){
    EZS[index(i,j,k,NX,NY,NZ)] = -EZSX2[index(1,j,k,4,NY1,NZ1)]+cxd*(EZS[index(1,j,k,NX,NY,NZ)]+EZSX2[index(i,j,k,4,NY1,NZ1)])+cxx*(EZSX1[index(i,j,k,4,NY1,NZ1)]+EZSX1[index(1,j,k,4,NY1,NZ1)])+cxfyd*(EZSX1[index(i,j+1,k,4,NY1,NZ1)]-2.0*EZSX1[index(i,j,k,4,NY1,NZ1)]+EZSX1[index(i,j-1,k,4,NY1,NZ1)]+EZSX1[index(1,j+1,k,4,NY1,NZ1)]-2.0*EZSX1[index(1,j,k,4,NY1,NZ1)]+EZSX1[index(1,j-1,k,4,NY1,NZ1)])+cxfzd*(EZSX1[index(i,j,k+1,4,NY1,NZ1)]-2.0*EZSX1[index(i,j,k,4,NY1,NZ1)]+EZSX1[index(i,j,k-1,4,NY1,NZ1)]+EZSX1[index(1,j,k+1,4,NY1,NZ1)]-2.0*EZSX1[index(1,j,k,4,NY1,NZ1)]+EZSX1[index(1,j,k-1,4,NY1,NZ1)]);
  }

  if(i==NX-1 && k>0 && k<NZ1-1 && j>1 && j<NY1-1){
    EZS[index(i,j,k,NX,NY,NZ)] = -EZSX2[index(2,j,k,4,NY1,NZ1)]+cxd*(EZS[index(NX1-1,j,k,NX,NY,NZ)]+EZSX2[index(3,j,k,4,NY1,NZ1)])+cxx*(EZSX1[index(3,j,k,4,NY1,NZ1)]+EZSX1[index(2,j,k,4,NY1,NZ1)])+cxfyd*(EZSX1[index(3,j+1,k,4,NY1,NZ1)]-2.0*EZSX1[index(3,j,k,4,NY1,NZ1)]+EZSX1[index(3,j-1,k,4,NY1,NZ1)]+EZSX1[index(2,j+1,k,4,NY1,NZ1)]-2.0*EZSX1[index(2,j,k,4,NY1,NZ1)]+EZSX1[index(2,j-1,k,4,NY1,NZ1)])+cxfzd*(EZSX1[index(3,j,k+1,4,NY1,NZ1)]-2.0*EZSX1[index(3,j,k,4,NY1,NZ1)]+EZSX1[index(3,j,k-1,4,NY1,NZ1)]+EZSX1[index(2,j,k+1,4,NY1,NZ1)]-2.0*EZSX1[index(2,j,k,4,NY1,NZ1)]+EZSX1[index(2,j,k-1,4,NY1,NZ1)]);
  }

}

__global__ void radezx_save(double *EZS, double *EZSX1, double *EZSX2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY1*NZ1))%NZ1;
  j=(tid%(NY1*NZ1))/NZ1;
  i=tid/(NY1*NZ1);

  if(k<NZ1 && j>0 && j<NY1){
    EZSX2[index(i,j,k,4,NY1,NZ1)]=EZSX1[index(i,j,k,4,NY1,NZ1)];
  }

  if((i==0 || i==1) && k<NZ1 && j>0 && j<NY1){
    EZSX1[index(i,j,k,4,NY1,NZ1)]=EZS[index(i,j,k,NX,NY,NZ)];
  }

  if(i==2 && k<NZ1 && j>0 && j<NY1){
    EZSX1[index(i,j,k,4,NY1,NZ1)]=EZS[index(NX1-1,j,k,NX,NY,NZ)];
  }

  if(i==3 && k<NZ1 && j>0 && j<NY1){
    EZSX1[index(i,j,k,4,NY1,NZ1)]=EZS[index(NX-1,j,k,NX,NY,NZ)];
  }
}

__global__ void radeyx1(double *EYS, double *EYSX1, double *EYSX2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(i==0 && k>0 && k<NZ1){
    j = 0;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EYS[index(1,j,k,NX,NY,NZ)] - EYSX1[index(i,j,k,4,NY1,NZ1)]);
    j = NY1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EYS[index(1,j,k,NX,NY,NZ)] - EYSX1[index(i,j,k,4,NY1,NZ1)]);
  }

  if(i==NX-1 && k>0 && k<NZ1){
    j = 0;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EYS[index(NX1-1,j,k,NX,NY,NZ)] - EYSX1[index(3,j,k,4,NY1,NZ1)]);
    j = NY1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EYS[index(NX1-1,j,k,NX,NY,NZ)] - EYSX1[index(3,j,k,4,NY1,NZ1)]);
  }

  if(i==0 && j>0 && j<NY1-1){
    k = 1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EYS[index(1,j,k,NX,NY,NZ)] - EYSX1[index(0,j,k,4,NY1,NZ1)]);
    k = NZ1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(1,j,k,4,NY1,NZ1)]+ cxd*(EYS[index(1,j,k,NX,NY,NZ)] - EYSX1[index(0,j,k,4,NY1,NZ1)]);
  }

  if(i==NX-1 && j>0 && j<NY1-1){
    k = 1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EYS[index(NX1-1,j,k,NX,NY,NZ)] - EYSX1[index(3,j,k,4,NY1,NZ1)]);
    k = NZ1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSX1[index(2,j,k,4,NY1,NZ1)] + cxu*(EYS[index(NX1-1,j,k,NX,NY,NZ)] - EYSX1[index(3,j,k,4,NY1,NZ1)]);
  }
  
}

__global__ void radeyx2(double *EYS, double *EYSX1, double *EYSX2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(i==0 && k>1 && k<NZ1-1 && j>0 && j<NY1-1){
    EYS[index(i,j,k,NX,NY,NZ)] = -EYSX2[index(1,j,k,4,NY1,NZ1)]+cxd*(EYS[index(1,j,k,NX,NY,NZ)]+EYSX2[index(i,j,k,4,NY1,NZ1)])+cxx*(EYSX1[index(i,j,k,4,NY1,NZ1)]+EYSX1[index(1,j,k,4,NY1,NZ1)])+cxfyd*(EYSX1[index(i,j+1,k,4,NY1,NZ1)]-2.0*EYSX1[index(i,j,k,4,NY1,NZ1)]+EYSX1[index(i,j-1,k,4,NY1,NZ1)]+EYSX1[index(1,j+1,k,4,NY1,NZ1)]-2.0*EYSX1[index(1,j,k,4,NY1,NZ1)]+EYSX1[index(1,j-1,k,4,NY1,NZ1)])+cxfzd*(EYSX1[index(i,j,k+1,4,NY1,NZ1)]-2.0*EYSX1[index(i,j,k,4,NY1,NZ1)]+EYSX1[index(i,j,k-1,4,NY1,NZ1)]+EYSX1[index(1,j,k+1,4,NY1,NZ1)]-2.0*EYSX1[index(1,j,k,4,NY1,NZ1)]+EYSX1[index(1,j,k-1,4,NY1,NZ1)]);
  }

  if(i==NX-1 && k>1 && k<NZ1-1 && j>0 && j<NY1-1){
    EYS[index(i,j,k,NX,NY,NZ)] = -EYSX2[index(2,j,k,4,NY1,NZ1)]+cxd*(EYS[index(NX1-1,j,k,NX,NY,NZ)]+EYSX2[index(3,j,k,4,NY1,NZ1)])+cxx*(EYSX1[index(3,j,k,4,NY1,NZ1)]+EYSX1[index(2,j,k,4,NY1,NZ1)])+cxfyd*(EYSX1[index(3,j+1,k,4,NY1,NZ1)]-2.0*EYSX1[index(3,j,k,4,NY1,NZ1)]+EYSX1[index(3,j-1,k,4,NY1,NZ1)]+EYSX1[index(2,j+1,k,4,NY1,NZ1)]-2.0*EYSX1[index(2,j,k,4,NY1,NZ1)]+EYSX1[index(2,j-1,k,4,NY1,NZ1)])+cxfzd*(EYSX1[index(3,j,k+1,4,NY1,NZ1)]-2.0*EYSX1[index(3,j,k,4,NY1,NZ1)]+EYSX1[index(3,j,k-1,4,NY1,NZ1)]+EYSX1[index(2,j,k+1,4,NY1,NZ1)]-2.0*EYSX1[index(2,j,k,4,NY1,NZ1)]+EYSX1[index(2,j,k-1,4,NY1,NZ1)]);
  }
  
}

__global__ void radeyx_save(double *EYS, double *EYSX1, double *EYSX2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY1*NZ1))%NZ1;
  j=(tid%(NY1*NZ1))/NZ1;
  i=tid/(NY1*NZ1);

  if(k>0 && k<NZ1 && j<NY1){
    EYSX2[index(i,j,k,4,NY1,NZ1)]=EYSX1[index(i,j,k,4,NY1,NZ1)];
  }

  if((i==0 || i==1) && k>0 && k<NZ1 && j<NY1){
    EYSX1[index(i,j,k,4,NY1,NZ1)]=EYS[index(i,j,k,NX,NY,NZ)];
  }

  if(i==2 && k>0 && k<NZ1 && j<NY1){
    EYSX1[index(i,j,k,4,NY1,NZ1)]=EYS[index(NX1-1,j,k,NX,NY,NZ)];
  }

  if(i==3 && k>0 && k<NZ1 && j<NY1){
    EYSX1[index(i,j,k,4,NY1,NZ1)]=EYS[index(NX-1,j,k,NX,NY,NZ)];
  }

}

__global__ void radezy1(double *EZS, double *EZSY1, double *EZSY2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(j==0 && k<NZ1){
    i = 1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EZS[index(i,1,k,NX,NY,NZ)] - EZSY1[index(i,j,k,NX1,4,NZ1)]);
    i = NX1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EZS[index(i,1,k,NX,NY,NZ)] - EZSY1[index(i,j,k,NX1,4,NZ1)]);
  }

  if(j==(NY-1) && k<NZ1){
    i = 1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EZS[index(i,NY1-1,k,NX,NY,NZ)] - EZSY1[index(i,3,k,NX1,4,NZ1)]);
    i = NX1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EZS[index(i,NY1-1,k,NX,NY,NZ)] - EZSY1[index(i,3,k,NX1,4,NZ1)]);
  }

  
  if(j==0 && i>1 && i<NX1-1){
    k = 0;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EZS[index(i,1,k,NX,NY,NZ)] - EZSY1[index(i,j,k,NX1,4,NZ1)]);
    k = NZ1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EZS[index(i,1,k,NX,NY,NZ)] - EZSY1[index(i,j,k,NX1,4,NZ1)]);
  }

  if(j==(NY-1) && i>1 && i<NX1-1){
    k = 0;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EZS[index(i,NY1-1,k,NX,NY,NZ)] - EZSY1[index(i,3,k,NX1,4,NZ1)]);
    k = NZ1-1;
    EZS[index(i,j,k,NX,NY,NZ)] = EZSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EZS[index(i,NY1-1,k,NX,NY,NZ)] - EZSY1[index(i,3,k,NX1,4,NZ1)]);
  }
      
}

__global__ void radezy2(double *EZS, double *EZSY1, double *EZSY2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(j==0 && k>0 && k<NZ1-1 && i>1 && i<NX1-1){
    
    EZS[index(i,j,k,NX,NY,NZ)] = -EZSY2[index(i,1,k,NX1,4,NZ1)]+cyd*(EZS[index(i,1,k,NX,NY,NZ)]+EZSY2[index(i,j,k,NX1,4,NZ1)])+cyy*(EZSY1[index(i,j,k,NX1,4,NZ1)]+EZSY1[index(i,1,k,NX1,4,NZ1)])+cyfxd*(EZSY1[index(i+1,j,k,NX1,4,NZ1)]-2.0*EZSY1[index(i,j,k,NX1,4,NZ1)]+EZSY1[index(i-1,j,k,NX1,4,NZ1)]+EZSY1[index(i+1,1,k,NX1,4,NZ1)]-2.0*EZSY1[index(i,1,k,NX1,4,NZ1)]+EZSY1[index(i-1,1,k,NX1,4,NZ1)])+cyfzd*(EZSY1[index(i,j,k+1,NX1,4,NZ1)]-2.0*EZSY1[index(i,j,k,NX1,4,NZ1)]+EZSY1[index(i,j,k-1,NX1,4,NZ1)]+EZSY1[index(i,1,k+1,NX1,4,NZ1)]-2.0*EZSY1[index(i,1,k,NX1,4,NZ1)]+EZSY1[index(i,1,k-1,NX1,4,NZ1)]);
  }
  
  if(j==(NY-1) && k>0 && k<NZ1-1 && i>1 && i<NX1-1){
    EZS[index(i,j,k,NX,NY,NZ)] = -EZSY2[index(i,2,k,NX1,4,NZ1)]+cyd*(EZS[index(i,NY1-1,k,NX,NY,NZ)]+EZSY2[index(i,3,k,NX1,4,NZ1)])+cyy*(EZSY1[index(i,3,k,NX1,4,NZ1)]+EZSY1[index(i,2,k,NX1,4,NZ1)])+cyfxd*(EZSY1[index(i+1,3,k,NX1,4,NZ1)]-2.0*EZSY1[index(i,3,k,NX1,4,NZ1)]+EZSY1[index(i-1,3,k,NX1,4,NZ1)]+EZSY1[index(i+1,2,k,NX1,4,NZ1)]-2.0*EZSY1[index(i,2,k,NX1,4,NZ1)]+EZSY1[index(i-1,2,k,NX1,4,NZ1)])+ cyfzd*(EZSY1[index(i,3,k+1,NX1,4,NZ1)]-2.0*EZSY1[index(i,3,k,NX1,4,NZ1)]+EZSY1[index(i,3,k-1,NX1,4,NZ1)]+EZSY1[index(i,2,k+1,NX1,4,NZ1)]-2.0*EZSY1[index(i,2,k,NX1,4,NZ1)]+EZSY1[index(i,2,k-1,NX1,4,NZ1)]);
  }
}

__global__ void radezy_save(double *EZS, double *EZSY1, double *EZSY2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(4*NZ1))%NZ1;
  j=(tid%(4*NZ1))/NZ1;
  i=tid/(4*NZ1);

  if(k<NZ1 && i>0 && i<NX1){
    EZSY2[index(i,j,k,NX1,4,NZ1)]=EZSY1[index(i,j,k,NX1,4,NZ1)];
  }

  if((j==0 || j==1) && k<NZ1 && i>0 && i<NX1){
    EZSY1[index(i,j,k,NX1,4,NZ1)]=EZS[index(i,j,k,NX,NY,NZ)];
  }

  if(j==2 && k<NZ1 && i>0 && i<NX1){
    EZSY1[index(i,j,k,NX1,4,NZ1)]=EZS[index(i,NY1-1,k,NX,NY,NZ)];
  }

  if(j==3 && k<NZ1 && i>0 && i<NX1){
    EZSY1[index(i,j,k,NX1,4,NZ1)]=EZS[index(i,NY-1,k,NX,NY,NZ)];
  }

}

__global__ void radexy1(double *EXS, double *EXSY1, double *EXSY2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(j==0 && k>0 && k<NZ1){
    i = 0;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EXS[index(i,1,k,NX,NY,NZ)] - EXSY1[index(i,j,k,NX1,4,NZ1)]);
    i = NX1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EXS[index(i,1,k,NX,NY,NZ)] - EXSY1[index(i,j,k,NX1,4,NZ1)]);
  }

  if(j==(NY-1) && k>0 && k<NZ1){
    i = 0;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EXS[index(i,NY1-1,k,NX,NY,NZ)] - EXSY1[index(i,3,k,NX1,4,NZ1)]);
    i = NX1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EXS[index(i,NY1-1,k,NX,NY,NZ)] - EXSY1[index(i,3,k,NX1,4,NZ1)]);
  }

  
  if(j==0 && i>0 && i<NX1-1){
    
    k = 1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EXS[index(i,1,k,NX,NY,NZ)] - EXSY1[index(i,j,k,NX1,4,NZ1)]);
    k = NZ1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,1,k,NX1,4,NZ1)]+ cyd*(EXS[index(i,1,k,NX,NY,NZ)] - EXSY1[index(i,j,k,NX1,4,NZ1)]);
  }

  if(j==(NY-1) && i>0 && i<NX1-1){
    k = 1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EXS[index(i,NY1-1,k,NX,NY,NZ)] - EXSY1[index(i,3,k,NX1,4,NZ1)]);
    k = NZ1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSY1[index(i,2,k,NX1,4,NZ1)] + cyd*(EXS[index(i,NY1-1,k,NX,NY,NZ)] - EXSY1[index(i,3,k,NX1,3,NZ1)]);
  }
}

__global__ void radexy2(double *EXS, double *EXSY1, double *EXSY2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(j==0 && k>1 && k<NZ1-1 && i>0 && i<NX1-1){
    EXS[index(i,j,k,NX,NY,NZ)] = -EXSY2[index(i,1,k,NX1,4,NZ1)]+cyd*(EXS[index(i,1,k,NX,NY,NZ)]+EXSY2[index(i,j,k,NX1,4,NZ1)])+cyy*(EXSY1[index(i,j,k,NX1,4,NZ1)]+EXSY1[index(i,1,k,NX1,4,NZ1)])+cyfxd*(EXSY1[index(i+1,j,k,NX1,4,NZ1)]-2.0*EXSY1[index(i,j,k,NX1,4,NZ1)]+EXSY1[index(i-1,j,k,NX1,4,NZ1)]+EXSY1[index(i+1,1,k,NX1,4,NZ1)]-2.0*EXSY1[index(i,1,k,NX1,4,NZ1)]+EXSY1[index(i-1,1,k,NX1,4,NZ1)])+cyfzd*(EXSY1[index(i,j,k+1,NX1,4,NZ1)]-2.0*EXSY1[index(i,j,k,NX1,4,NZ1)]+EXSY1[index(i,j,k-1,NX1,4,NZ1)]+EXSY1[index(i,1,k+1,NX1,4,NZ1)]-2.0*EXSY1[index(i,1,k,NX1,4,NZ1)]+EXSY1[index(i,1,k-1,NX1,4,NZ1)]);
  }

  if(j==(NY-1) && k>1 && k<NZ1-1 && i>0 && i<NX1-1){
    EXS[index(i,j,k,NX,NY,NZ)] = -EXSY2[index(i,2,k,NX1,4,NZ1)]+cyd*(EXS[index(i,NY1-1,k,NX,NY,NZ)]+EXSY2[index(i,3,k,NX1,4,NZ1)])+cyy*(EXSY1[index(i,3,k,NX1,4,NZ1)]+EXSY1[index(i,2,k,NX1,4,NZ1)])+cyfxd*(EXSY1[index(i+1,3,k,NX1,4,NZ1)]-2.0*EXSY1[index(i,3,k,NX1,4,NZ1)]+EXSY1[index(i-1,3,k,NX1,4,NZ1)]+EXSY1[index(i+1,2,k,NX1,4,NZ1)]-2.0*EXSY1[index(i,2,k,NX1,4,NZ1)]+EXSY1[index(i-1,2,k,NX1,4,NZ1)])+cyfzd*(EXSY1[index(i,3,k+1,NX1,4,NZ1)]-2.0*EXSY1[index(i,3,k,NX1,4,NZ1)]+EXSY1[index(i,3,k-1,NX1,4,NZ1)]+EXSY1[index(i,2,k+1,NX1,4,NZ1)]-2.0*EXSY1[index(i,2,k,NX1,4,NZ1)]+EXSY1[index(i,2,k-1,NX1,4,NZ1)]);
  }
}

__global__ void radexy_save(double *EXS, double *EXSY1, double *EXSY2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(4*NZ1))%NZ1;
  j=(tid%(4*NZ1))/NZ1;
  i=tid/(4*NZ1);

  if(k>0 && k<NZ1 && i<NX1){
    EXSY2[index(i,j,k,NX1,4,NZ1)]=EXSY1[index(i,j,k,NX1,4,NZ1)];
  }

  if((j==0 || j==1) && k>0 && k<NZ1 && i<NX1){
    EXSY1[index(i,j,k,NX1,4,NZ1)]=EXS[index(i,j,k,NX,NY,NZ)];
  }

  if(j==2 && k>0 && k<NZ1 && i<NX1){
    EXSY1[index(i,j,k,NX1,4,NZ1)]=EXS[index(i,NY1-1,k,NX,NY,NZ)];
  }

  if(j==3 && k>0 && k<NZ1 && i<NX1){
    EXSY1[index(i,j,k,NX1,4,NZ1)]=EXS[index(i,NY-1,k,NX,NY,NZ)];
  }
}

__global__ void radexz1(double *EXS, double *EXSZ1, double *EXSZ2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(k==0 && j>0 && j<NY1){
    i = 0;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EXS[index(i,j,1,NX,NY,NZ)] - EXSZ1[index(i,j,k,NX1,NY1,4)]);
    i = NX1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EXS[index(i,j,1,NX,NY,NZ)] - EXSZ1[index(i,j,k,NX1,NY1,4)]);
  }

  if(k==(NZ-1) && j>0 && j<NY1){
    i = 0;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EXS[index(i,j,NZ1-1,NX,NY,NZ)] - EXSZ1[index(i,j,3,NX1,NY1,4)]);
    i = NX1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EXS[index(i,j,NZ1-1,NX,NY,NZ)] - EXSZ1[index(i,j,3,NX1,NY1,4)]);
  }

  if(k==0 && i>0 && i<NX1-1){
    j = 1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EXS[index(i,j,1,NX,NY,NZ)] - EXSZ1[index(i,j,k,NX1,NY1,4)]);
    j = NY1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EXS[index(i,j,1,NX,NY,NZ)] - EXSZ1[index(i,j,k,NX1,NY1,4)]);
  }

  if(k==(NZ-1) && i>0 && i<NX1-1){
    j = 1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EXS[index(i,j,NZ1-1,NX,NY,NZ)] - EXSZ1[index(i,j,3,NX1,NY1,4)]);
    j = NY1-1;
    EXS[index(i,j,k,NX,NY,NZ)] = EXSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EXS[index(i,j,NZ1-1,NX,NY,NZ)] - EXSZ1[index(i,j,3,NX1,NY1,4)]);
  }
}

__global__ void radexz2(double *EXS, double *EXSZ1, double *EXSZ2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(k==0 && j>1 && j<NY1-1 && i>0 && i<NX1-1){
    EXS[index(i,j,k,NX,NY,NZ)] = -EXSZ2[index(i,j,1,NX1,NY1,4)]+czd*(EXS[index(i,j,1,NX,NY,NZ)]+EXSZ2[index(i,j,k,NX1,NY1,4)])+czz*(EXSZ1[index(i,j,k,NX1,NY1,4)]+EXSZ1[index(i,j,1,NX1,NY1,4)])+czfxd*(EXSZ1[index(i+1,j,k,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,k,NX1,NY1,4)]+EXSZ1[index(i-1,j,k,NX1,NY1,4)]+EXSZ1[index(i+1,j,1,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,1,NX1,NY1,4)]+EXSZ1[index(i-1,j,1,NX1,NY1,4)])+czfyd*(EXSZ1[index(i,j+1,k,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,k,NX1,NY1,4)]+EXSZ1[index(i,j-1,k,NX1,NY1,4)]+EXSZ1[index(i,j+1,1,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,1,NX1,NY1,4)]+EXSZ1[index(i,j-1,1,NX1,NY1,4)]);
  }

  if(k==(NZ-1) && j>1 && j<NY1-1 && i>0 && i<NX1-1){
    EXS[index(i,j,k,NX,NY,NZ)] = -EXSZ2[index(i,j,2,NX1,NY1,4)]+czd*(EXS[index(i,j,NZ1-1,NX,NY,NZ)]+EXSZ2[index(i,j,3,NX1,NY1,4)])+czz*(EXSZ1[index(i,j,3,NX1,NY1,4)]+EXSZ1[index(i,j,2,NX1,NY1,4)])+czfxd*(EXSZ1[index(i+1,j,3,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,3,NX1,NY1,4)]+EXSZ1[index(i-1,j,3,NX1,NY1,4)]+EXSZ1[index(i+1,j,2,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,2,NX1,NY1,4)]+EXSZ1[index(i-1,j,2,NX1,NY1,4)])+czfyd*(EXSZ1[index(i,j+1,3,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,3,NX1,NY1,4)]+EXSZ1[index(i,j-1,3,NX1,NY1,4)]+EXSZ1[index(i,j+1,2,NX1,NY1,4)]-2.0*EXSZ1[index(i,j,2,NX1,NY1,4)]+EXSZ1[index(i,j-1,2,NX1,NY1,4)]);
  }
}

__global__ void radexz_save(double *EXS, double *EXSZ1, double *EXSZ2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY1*4))%4;
  j=(tid%(NY1*4))/4;
  i=tid/(NY1*4);

  if(j>0 && j<NY1 && i<NX1){
    EXSZ2[index(i,j,k,NX1,NY1,4)]=EXSZ1[index(i,j,k,NX1,NY1,4)];
  }

  if((k==0 || k==1) && j>0 && j<NY1 && i<NX1){
    EXSZ1[index(i,j,k,NX1,NY1,4)]=EXS[index(i,j,k,NX,NY,NZ)];
  }

  if(k==2 && j>0 && j<NY1 && i<NX1){
    EXSZ1[index(i,j,k,NX1,NY1,4)]=EXS[index(i,j,NZ1-1,NX,NY,NZ)];
  }

  if(k==3 && j>0 && j<NY1 && i<NX1){
    EXSZ1[index(i,j,k,NX1,NY1,4)]=EXS[index(i,j,NZ-1,NX,NY,NZ)];
  }
}

__global__ void radeyz1(double *EYS, double *EYSZ1, double *EYSZ2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(k==0 && j<NY1){
    i = 1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EYS[index(i,j,1,NX,NY,NZ)] - EYSZ1[index(i,j,k,NX1,NY1,4)]);
    i = NX1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EYS[index(i,j,1,NX,NY,NZ)] - EYSZ1[index(i,j,k,NX1,NY1,4)]);
  }

  if(k==(NZ-1) && j<NY1){
    i = 1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EYS[index(i,j,NZ1-1,NX,NY,NZ)] - EYSZ1[index(i,j,3,NX1,NY1,4)]);
    i = NX1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EYS[index(i,j,NZ1-1,NX,NY,NZ)] - EYSZ1[index(i,j,3,NX1,NY1,4)]);
  }

  if(k==0 && i>1 && i<NX1-1){
    j = 0;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EYS[index(i,j,1,NX,NY,NZ)] - EYSZ1[index(i,j,k,NX1,NY1,4)]);
    j = NY1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,1,NX1,NY1,4)]+ czd*(EYS[index(i,j,1,NX,NY,NZ)] - EYSZ1[index(i,j,k,NX1,NY1,4)]);
  }

  if(k==(NZ-1) && i>1 && i<NX1-1){
    j = 0;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EYS[index(i,j,NZ1-1,NX,NY,NZ)] - EYSZ1[index(i,j,3,NX1,NY1,4)]);
    j = NY1-1;
    EYS[index(i,j,k,NX,NY,NZ)] = EYSZ1[index(i,j,2,NX1,NY1,4)] + czd*(EYS[index(i,j,NZ1-1,NX,NY,NZ)] - EYSZ1[index(i,j,3,NX1,NY1,4)]);
  }
}

__global__ void radeyz2(double *EYS, double *EYSZ1, double *EYSZ2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(k==0 && j>0 && j<NY1-1 && i>1 && i<NX1-1){
    EYS[index(i,j,k,NX,NY,NZ)] = -EYSZ2[index(i,j,1,NX1,NY1,4)]+ czd*(EYS[index(i,j,1,NX,NY,NZ)]+EYSZ2[index(i,j,k,NX1,NY1,4)])+czz*(EYSZ1[index(i,j,k,NX1,NY1,4)]+EYSZ1[index(i,j,1,NX1,NY1,4)])+ czfxd*(EYSZ1[index(i+1,j,k,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,k,NX1,NY1,4)]+EYSZ1[index(i-1,j,k,NX1,NY1,4)]+EYSZ1[index(i+1,j,1,NX1,NY1,4)]- 2.0*EYSZ1[index(i,j,1,NX1,NY1,4)]+EYSZ1[index(i-1,j,1,NX1,NY1,4)])+ czfyd*(EYSZ1[index(i,j+1,k,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,k,NX1,NY1,4)]+EYSZ1[index(i,j-1,k,NX1,NY1,4)]+EYSZ1[index(i,j+1,1,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,1,NX1,NY1,4)]+EYSZ1[index(i,j-1,1,NX1,NY1,4)]);
  }

  if(k==(NZ-1) && j>0 && j<NY1-1 && i>1 && i<NX1-1){
    EYS[index(i,j,k,NX,NY,NZ)] = -EYSZ2[index(i,j,2,NX1,NY1,4)]+czd*(EYS[index(i,j,NZ1-1,NX,NY,NZ)]+EYSZ2[index(i,j,3,NX1,NY1,4)])+czz*(EYSZ1[index(i,j,3,NX1,NY1,4)]+EYSZ1[index(i,j,2,NX1,NY1,4)])+czfxd*(EYSZ1[index(i+1,j,3,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,3,NX1,NY1,4)]+EYSZ1[index(i-1,j,3,NX1,NY1,4)]+EYSZ1[index(i+1,j,2,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,2,NX1,NY1,4)]+EYSZ1[index(i-1,j,2,NX1,NY1,4)])+czfyd*(EYSZ1[index(i,j+1,3,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,3,NX1,NY1,4)]+EYSZ1[index(i,j-1,3,NX1,NY1,4)]+EYSZ1[index(i,j+1,2,NX1,NY1,4)]-2.0*EYSZ1[index(i,j,2,NX1,NY1,4)]+EYSZ1[index(i,j-1,2,NX1,NY1,4)]);
  }
}

__global__ void radeyz_save(double *EYS, double *EYSZ1, double *EYSZ2, int NX, int NY, int NZ, int NX1, int NY1, int NZ1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY1*4))%4;
  j=(tid%(NY1*4))/4;
  i=tid/(NY1*4);

  if(j<NY1 && i>0 && i<NX1){
    EYSZ2[index(i,j,k,NX1,NY1,4)]=EYSZ1[index(i,j,k,NX1,NY1,4)];
  }

  if((k==0 || k==1) && j<NY1 && i>0 && i<NX1){
    EYSZ1[index(i,j,k,NX1,NY1,4)]=EYS[index(i,j,k,NX,NY,NZ)];
  }

  if(k==2 && j<NY1 && i>0 && i<NX1){
    EYSZ1[index(i,j,k,NX1,NY1,4)]=EYS[index(i,j,NZ1-1,NX,NY,NZ)];
  }

  if(k==3 && j<NY1 && i>0 && i<NX1){
    EYSZ1[index(i,j,k,NX1,NY1,4)]=EYS[index(i,j,NZ-1,NX,NY,NZ)];
  }
}

__global__ void hupdate(double *EXS, double *EYS, double *EZS, double *HXS, double *HYS, double *HZS, int NZ1, int NY1, int NX1){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(i>0 && i<NX1 && j<NY1 && k<NZ1){
    HXS[tid]=HXS[tid]-(EZS[index(i,j+1,k,NX,NY,NZ)]-EZS[tid])*dtmdy+(EYS[index(i,j,k+1,NX,NY,NZ)]-EYS[tid])*dtmdz;
  }

  if(i<NX1 && j>0 && j<NY1 && k<NZ1){
    HYS[tid]=HYS[tid]-(EXS[index(i,j,k+1,NX,NY,NZ)]-EXS[tid])*dtmdz+(EZS[index(i+1,j,k,NX,NY,NZ)]-EZS[tid])*dtmdx;
  }

  if(i<NX1 && j<NY1 && k>0 && k<NZ1){
    HZS[tid]=HZS[tid]-(EYS[index(i+1,j,k,NX,NY,NZ)]-EYS[tid])*dtmdx+(EXS[index(i,j+1,k,NX,NY,NZ)]-EXS[tid])*dtmdy;
  }
}

__global__ void datsav1(double *etimeavg, double *emax, double *EXS, double *EYS, double *EZS, int *count, int N, double t, double *dt, int NX, int NY, int NZ){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  double nrms,exc=0.0,eyc=0.0,ezc=0.0,exin,eyin,ezin,esq=0.0;
  nrms=(1.0/(freq*(*dt)));
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);

  if(i<NX-2 && j<NY-2 && k<NZ-2){
    
    t=t-(*dt);
    exin=EXI(i,j,k,t)+EXI(i,j,k+1,t)+EXI(i,j+1,k,t)+EXI(i,j+1,k+1,t);
    eyin=EYI(i,j,k,t)+EYI(i+1,j,k,t)+EYI(i,j,k+1,t)+EYI(i+1,j,k+1,t);
    ezin=EZI(i,j,k,t)+EZI(i+1,j,k,t)+EZI(i+1,j+1,k,t)+EZI(i,j+1,k,t);
    t=t+(*dt);
    
    exc=(exin+EXS[index(i,j,k,NX,NY,NZ)]+EXS[index(i,j,k+1,NX,NY,NZ)]+EXS[index(i,j+1,k,NX,NY,NZ)]+EXS[index(i,j+1,k+1,NX,NY,NZ)])*(0.25);
    eyc=(eyin+EYS[index(i,j,k,NX,NY,NZ)]+EYS[index(i+1,j,k,NX,NY,NZ)]+EYS[index(i,j,k+1,NX,NY,NZ)]+EYS[index(i+1,j,k+1,NX,NY,NZ)])*(0.25);
    ezc=(ezin+EZS[index(i,j,k,NX,NY,NZ)]+EZS[index(i+1,j,k,NX,NY,NZ)]+EZS[index(i+1,j+1,k,NX,NY,NZ)]+EZS[index(i,j+1,k,NX,NY,NZ)])*(0.25);
    
    esq=((exc*exc)+(eyc*eyc)+(ezc*ezc));
    (etimeavg[index(i,j,k,NX,NY,NZ)])=(etimeavg[index(i,j,k,NX,NY,NZ)])+(esq/nrms);
    
    if(N==(nstop-nrms)) emax[index(i,j,k,NX,NY,NZ)]=esq;
    if(esq>(emax[index(i,j,k,NX,NY,NZ)])) emax[index(i,j,k,NX,NY,NZ)]=esq;
  }
  return;
}

__global__ void sar_cal(double *etimeavg, double *emax, int *IDONE, double *count2, double *sarx, int NX, int NY, int NZ){

  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  int i,j,k;
  k=(tid%(NY*NZ))%NZ;
  j=(tid%(NY*NZ))/NZ;
  i=tid/(NY*NZ);
  double temp;
  double r2=0.5, r3=0.1;

  if(wbsar==1 && i<NX-2 && j<NY-2 && k<NZ-2){
    temp=((pow((i-nxc)*delx,2)/pow(r3,2)) + (pow((j-nyc)*dely,2)/pow(r3,2)) + (pow((k-nzc)*delz,2)/pow(r2,2)));

    // temp=((((i-nxc)*delx)*((i-nxc)*delx))+(((j-nyc)*dely)*((j-nyc)*dely))+(((k-nzc)*delz)*((k-nzc)*delz)));
    // r1=sqrt(temp);
    
    if(/*r1<=radius2*/ temp>0.0 && temp<=1.0){
      count2[tid]=1;
      sarx[tid]=((0.5)*SIGMA[IDONE[index(i,j,k,NX,NY,NZ)]-1]*(emax[index(i,j,k,NX,NY,NZ)])*(delx*dely*delz));
      // printf("%.25lf\n",sarx[tid]);
    }
  }
}


int main(){
  
  // double *EXS, *EYS, *EZS;
  // double *HXS, *HYS, *HZS;
  // double *EXSY1, *EXSY2, *EXSZ1, *EXSZ2, *EYSX1, *EYSX2, *EYSZ1, *EYSZ2, *EZSX1, *EZSX2, *EZSY1, *EZSY2;
  // int *IDONE, *IDTWO, *IDTHREE;
  // double *EZS;
  // int *IDONE;

  double *d_EXS, *d_EYS, *d_EZS;
  double *d_HXS, *d_HYS, *d_HZS;
  double *d_EXSY1, *d_EXSY2, *d_EXSZ1, *d_EXSZ2, *d_EYSX1, *d_EYSX2, *d_EYSZ1, *d_EYSZ2, *d_EZSX1, *d_EZSX2, *d_EZSY1, *d_EZSY2;
  int *d_IDONE, *d_IDTWO, *d_IDTHREE;

  long int domain_size=NX*NY*NZ;
  //double *emax;
  int *d_count;
  double *d_etimeavg, *d_emax;// , *d_s, *d_erms1d, *d_erms, *d_sar,

  // EXS=(double*)malloc(domain_size*sizeof(double));
  // EYS=(double*)malloc(domain_size*sizeof(double));
  // EZS=(double*)malloc(domain_size*sizeof(double));
  // HXS=(double*)malloc(domain_size*sizeof(double));
  // HYS=(double*)malloc(domain_size*sizeof(double));
  // HZS=(double*)malloc(domain_size*sizeof(double));
  // EXSY1=(double*)malloc(NX1*4*NZ1*sizeof(double));
  // EXSY2=(double*)malloc(NX1*4*NZ1*sizeof(double));
  // EXSZ1=(double*)malloc(NX1*NY1*4*sizeof(double));
  // EXSZ2=(double*)malloc(NX1*NY1*4*sizeof(double));
  // EYSX1=(double*)malloc(4*NY1*NZ1*sizeof(double));
  // EYSX2=(double*)malloc(4*NY1*NZ1*sizeof(double));
  // EYSZ1=(double*)malloc(NX1*NY1*4*sizeof(double));
  // EYSZ2=(double*)malloc(NX1*NY1*4*sizeof(double));
  // EZSX1=(double*)malloc(4*NY1*NZ1*sizeof(double));
  // EZSX2=(double*)malloc(4*NY1*NZ1*sizeof(double));
  // EZSY1=(double*)malloc(NX1*4*NZ1*sizeof(double));
  // EZSY2=(double*)malloc(NX1*4*NZ1*sizeof(double));
  // IDONE=(int*)malloc(domain_size*sizeof(int));
  // IDTWO=(int*)malloc(domain_size*sizeof(int));
  // IDTHREE=(int*)malloc(domain_size*sizeof(int));
  // emax=(double*)malloc(domain_size*sizeof(double));
  
  hipMalloc((void **)&d_EXS, domain_size*sizeof(double));
  hipMalloc((void **)&d_EYS, domain_size*sizeof(double));
  hipMalloc((void **)&d_EZS, domain_size*sizeof(double));
  hipMalloc((void **)&d_HXS, domain_size*sizeof(double));
  hipMalloc((void **)&d_HYS, domain_size*sizeof(double));
  hipMalloc((void **)&d_HZS, domain_size*sizeof(double));
  hipMalloc((void **)&d_EXSY1, NX1*4*NZ1*sizeof(double));
  hipMalloc((void **)&d_EXSY2, NX1*4*NZ1*sizeof(double));
  hipMalloc((void **)&d_EXSZ1, NX1*NY1*4*sizeof(double));
  hipMalloc((void **)&d_EXSZ2, NX1*NY1*4*sizeof(double));
  hipMalloc((void **)&d_EYSX1, 4*NY1*NZ1*sizeof(double));
  hipMalloc((void **)&d_EYSX2, 4*NY1*NZ1*sizeof(double));
  hipMalloc((void **)&d_EYSZ1, NX1*NY1*4*sizeof(double));
  hipMalloc((void **)&d_EYSZ2, NX1*NY1*4*sizeof(double));
  hipMalloc((void **)&d_EZSX1, 4*NY1*NZ1*sizeof(double));
  hipMalloc((void **)&d_EZSX2, 4*NY1*NZ1*sizeof(double));
  hipMalloc((void **)&d_EZSY1, NX1*4*NZ1*sizeof(double));
  hipMalloc((void **)&d_EZSY2, NX1*4*NZ1*sizeof(double));
  hipMalloc((void **)&d_IDONE, domain_size*sizeof(int));
  hipMalloc((void **)&d_IDTWO, domain_size*sizeof(int));
  hipMalloc((void **)&d_IDTHREE, domain_size*sizeof(int));

  hipMalloc((void **)&d_etimeavg,domain_size*sizeof(double));
  hipMalloc((void **)&d_emax,domain_size*sizeof(double));

  double *d_dt;
  hipMalloc((void **)&d_dt, sizeof(double));

  long int blocks_per_grid;
  int threads_per_block=512;
  blocks_per_grid=(domain_size+threads_per_block-1)/threads_per_block;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  zero_fields<<<blocks_per_grid, threads_per_block>>>(d_EXS, d_EYS, d_EZS, d_HXS, d_HYS, d_HZS, d_IDONE, d_IDTWO, d_IDTHREE, domain_size);

  zero_xplanes<<<(4*NY1*NZ1+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EYSX1, d_EYSX2, d_EZSX1, d_EZSX2, 4*NY1*NZ1);
  
  zero_yplanes<<<(NX1*4*NZ1+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EXSY1, d_EXSY2, d_EZSY1, d_EZSY2, NX1*4*NZ1);

  zero_zplanes<<<(NX1*NY1*4+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EXSZ1, d_EXSZ2, d_EYSZ1, d_EYSZ2, NX1*NY1*4);

  zeromdim<<<1,1>>>();

  build<<<(domain_size+threads_per_block-1)/threads_per_block, threads_per_block>>>(NX, NY, NZ, d_IDONE, d_IDTWO, d_IDTHREE, domain_size);

  hipDeviceSynchronize();

  setup<<<1,1>>>(d_dt);

  hipDeviceSynchronize();
  
  /*FILE *fpe;
  fpe=fopen("ezs.txt","w");*/
  hipMemcpy(&dt, d_dt, sizeof(double), hipMemcpyDeviceToHost);
  //printf("dt : %.15lf\n",dt);
  /*FILE *fsar;
  fsar=fopen("emax.txt", "w");*/
  double nrms=(1.0/(frequency*dt));
  
  for(N=1;N<=nstop;N++){
    //printf("%d\n", N);
    //printf("%d\n",N);
    //k<<<blocks_per_grid, threads_per_block>>>(d_EXS, d_EYS, d_EZS, t);
    //hipDeviceSynchronize();
    //printf("kernel done\n");
    eupdate<<<blocks_per_grid, threads_per_block>>>(d_EXS, d_EYS, d_EZS, d_HXS, d_HYS, d_HZS, d_IDONE, d_IDTWO, d_IDTHREE, NZ1, NY1, NX1, t);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    //hipDeviceSynchronize();

    hipDeviceSynchronize();

  // check for error
  /*hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
    exit(0);*/
 
    radeyx1<<<blocks_per_grid,threads_per_block>>>(d_EYS, d_EYSX1, d_EYSX2, NX, NY, NZ, NX1, NY1, NZ1);
    //hipDeviceSynchronize();
    radeyx2<<<blocks_per_grid,threads_per_block>>>(d_EYS, d_EYSX1, d_EYSX2, NX, NY, NZ, NX1, NY1, NZ1);
    hipDeviceSynchronize();
    radeyx_save<<<((4*NY1*NZ1)+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EYS, d_EYSX1, d_EYSX2, NX, NY, NZ, NX1, NY1, NZ1);

    hipDeviceSynchronize();
    
    radezx1<<<blocks_per_grid,threads_per_block>>>(d_EZS, d_EZSX1, d_EZSX2, NX, NY, NZ, NX1, NY1, NZ1);
    //hipDeviceSynchronize();
    radezx2<<<blocks_per_grid,threads_per_block>>>(d_EZS, d_EZSX1, d_EZSX2, NX, NY, NZ, NX1, NY1, NZ1);
    hipDeviceSynchronize();
    radezx_save<<<((4*NY1*NZ1)+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EZS, d_EZSX1, d_EZSX2, NX, NY, NZ, NX1, NY1, NZ1);

    hipDeviceSynchronize();
    
    radezy1<<<blocks_per_grid,threads_per_block>>>(d_EZS, d_EZSY1, d_EZSY2, NX, NY, NZ, NX1, NY1, NZ1);
    radezy2<<<blocks_per_grid,threads_per_block>>>(d_EZS, d_EZSY1, d_EZSY2, NX, NY, NZ, NX1, NY1, NZ1);
    hipDeviceSynchronize();
    radezy_save<<<((NX1*4*NZ1)+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EZS, d_EZSY1, d_EZSY2, NX, NY, NZ, NX1, NY1, NZ1);

    hipDeviceSynchronize();
    
    radexy1<<<blocks_per_grid,threads_per_block>>>(d_EXS, d_EXSY1, d_EXSY2, NX, NY, NZ, NX1, NY1, NZ1);
    radexy2<<<blocks_per_grid,threads_per_block>>>(d_EXS, d_EXSY1, d_EXSY2, NX, NY, NZ, NX1, NY1, NZ1);
    hipDeviceSynchronize();
    radexy_save<<<((NX1*4*NZ1)+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EXS, d_EXSY1, d_EXSY2, NX, NY, NZ, NX1, NY1, NZ1);

    hipDeviceSynchronize();
    
    radexz1<<<blocks_per_grid,threads_per_block>>>(d_EXS, d_EXSZ1, d_EXSZ2, NX, NY, NZ, NX1, NY1, NZ1);
    radexz2<<<blocks_per_grid,threads_per_block>>>(d_EXS, d_EXSZ1, d_EXSZ2, NX, NY, NZ, NX1, NY1, NZ1);
    hipDeviceSynchronize();
    radexz_save<<<((NX1*NY1*4)+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EXS, d_EXSZ1, d_EXSZ2, NX, NY, NZ, NX1, NY1, NZ1);
	
    hipDeviceSynchronize();
    
    radeyz1<<<blocks_per_grid,threads_per_block>>>(d_EYS, d_EYSZ1, d_EYSZ2, NX, NY, NZ, NX1, NY1, NZ1);
    radeyz2<<<blocks_per_grid,threads_per_block>>>(d_EYS, d_EYSZ1, d_EYSZ2, NX, NY, NZ, NX1, NY1, NZ1);
    hipDeviceSynchronize();
    radeyz_save<<<((NX1*NY1*4)+threads_per_block-1)/threads_per_block,threads_per_block>>>(d_EYS, d_EYSZ1, d_EYSZ2, NX, NY, NZ, NX1, NY1, NZ1);

    hipDeviceSynchronize();
    
    t=t+((dt)/2.0);
    /*printf("t: %.15lf\n",t);
    hipMemcpy(EZS,d_EZS, domain_size*sizeof(double), hipMemcpyDeviceToHost);
    fprintf(fpe, "%.15lf %d %.25lf\n",t,N,EZS[index(50,50,50,NX,NY,NZ)]);
    */
    hupdate<<<blocks_per_grid, threads_per_block>>>(d_EXS, d_EYS, d_EZS, d_HXS, d_HYS, d_HZS, NZ1, NY1, NX1);

    hipDeviceSynchronize();

    t=t+((dt)/2.0);

    if(N>=(nstop-nrms) && N<=(nstop-1)){
      datsav1<<<blocks_per_grid, threads_per_block>>>(d_etimeavg, d_emax, d_EXS, d_EYS, d_EZS, d_count, N, t, d_dt, NX, NY, NZ);
      hipDeviceSynchronize();
      /*hipMemcpy(emax, d_emax, domain_size*sizeof(double),hipMemcpyDeviceToHost);
      fprintf(fsar, "%d %.15lf\n",N,emax[index(50,50,50,NX,NY,NZ)]);*/
    }
    hipDeviceSynchronize();

  }

  //fclose(fpe);
  /* 
  hipMemcpy(IDONE, d_IDONE, domain_size*sizeof(int), hipMemcpyDeviceToHost);
  printf("copying scatterer data\n");
  FILE *fp;
  fp=fopen("log.txt","w");
  int i,j,k;
  for(long int it=0;it<domain_size;it++){
    k=(it%(NY*NZ))%NZ;
    j=(it%(NY*NZ))/NZ;
    i=it/(NY*NZ);
    if(IDONE[it]==2)
    fprintf(fp,"%d %d %d %d\n", i,j,k,IDONE[it]);
  }
  fclose(fp);
  */
  //free(IDONE);
  //free(EZS);
  
  hipFree(d_EXS);
  hipFree(d_EYS);
  hipFree(d_EZS);
  hipFree(d_HXS);
  hipFree(d_HYS);
  hipFree(d_HZS);
  hipFree(d_EXSY1);
  hipFree(d_EXSY2);
  hipFree(d_EXSZ1);
  hipFree(d_EXSZ2);
  hipFree(d_EYSX1);
  hipFree(d_EYSX2);
  hipFree(d_EYSZ1);
  hipFree(d_EYSZ2);
  hipFree(d_EZSX1);
  hipFree(d_EZSX2);
  hipFree(d_EZSY1);
  hipFree(d_EXSY2);
  // hipFree(d_IDONE);
  hipFree(d_IDTWO);
  hipFree(d_IDTHREE);

  double *d_count2;
  double *d_sarx;

  hipMalloc((void **)&d_count2, domain_size*sizeof(double));
  hipMemset(d_count2,0,domain_size*sizeof(double));
  hipMalloc((void **)&d_sarx, domain_size*sizeof(double));
  hipMemset(d_sarx,0,domain_size*sizeof(double));
  
  sar_cal<<<blocks_per_grid, threads_per_block,threads_per_block*sizeof(double)>>>(d_etimeavg, d_emax, d_IDONE, d_count2, d_sarx, NX, NY, NZ);
  hipDeviceSynchronize();

  thrust::device_ptr<double> cptr1=thrust::device_pointer_cast(d_count2);
  double count2=thrust::reduce(cptr1,cptr1+domain_size);
  thrust::device_ptr<double> cptr2=thrust::device_pointer_cast(d_sarx);
  double sarx=thrust::reduce(cptr2,cptr2+domain_size);

  printf("Total cell count: %lf\ncells used for SAR calculation: %lf\nTotal power: %.15lf\nWhole Body SAR: %.15lf\n",count2, count2, sarx, (sarx)/(count2*h_delx*h_dely*h_delz*h_iden));

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float runtime=0;
  hipEventElapsedTime(&runtime, start, stop);
  printf("Run time: %lf\n",runtime);

  hipFree(d_count2);
  hipFree(d_sarx);  
  
  return 0;
}
